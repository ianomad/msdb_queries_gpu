#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/device_functions.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <sstream>

#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <cmath>

#include "types.h"
#include "utils.h"
#include "gpu_single.h"

#include "utils.cu"
#include "gpu_single.cu"


int block_size = 1024; //1025 max (# of threads in a block)

int main(int argc, char *argv[]) {

    /**
    * Read the number of particles
    */
    int numOfParticles = atoi(argv[1]);

    /**
    * Read name of the file
    */
    std::string fileName = argv[2];
    std::ifstream stream(fileName.c_str());
    std::cout << "Reading file: " << fileName << std::endl;

    atom* atomsList = new atom[numOfParticles];

    int heads = 0;
    int atomCount = 0;

    std::string token;
    std::string line;

    while(!stream.eof()) {
        //read line from file
        std::getline(stream, line);

        std::stringstream lineStream(line);
        
        lineStream >> token;
        if(token.compare("HEAD") == 0) {
            //skip the header

            std::cout << line << std::endl;

            heads++;
            std::cout << "Frame #" << heads << " processing. " << std::endl;
            std::cout << atomCount << " atoms read in previous frame." << std::endl;

            if(atomCount > 0) {
                run_single_kernel(atomsCnt, atomsList);
            }
            
            atomCount = 0;
            continue;
        }

        //example: `ATOM  00000000    00000001    00000001    17.297  15.357  5.428   -0.548  15.9994`
        //skip some stuff
        lineStream >> token;

        //std::cout << token << std::endl;

        lineStream >> token;
        lineStream >> token;

        //double x, y, z, charge, mass;
        lineStream >> atomsList[atomCount].x;
        lineStream >> atomsList[atomCount].y;
        lineStream >> atomsList[atomCount].z;
        lineStream >> atomsList[atomCount].charge;
        lineStream >> atomsList[atomCount].mass;

        atomCount++;
    }

    printf("Heads: %d\n", heads);
    printf("Atom Count: %d\n", atomCount);

    // run_single_kernel(atoms_cnt, workload);

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

	return 0;
}
