#include "hip/hip_runtime.h"
#include "gpu_single.h"
#include "types.h"


//1 body functions
__global__
void gpu_single_kernel(int* g_s_atomsCnt, atom* g_s_atom_list, query_results* g_s_res) {

    extern __shared__ int sdata[];

    int tid = threadIdx.x;

    if(tid >= *g_s_atomsCnt) {
        return;
    }
    
    int i = tid + blockDim.x;
    sdata[tid] = g_s_atom_list[tid].mass;

    while(i < *g_s_atomsCnt) {
        sdata[tid] += g_s_atom_list[i].mass;
        i += blockDim.x;
        __syncthreads();
    }

    atomicAdd(&g_s_res->mass, sdata[tid]);
}

void run_single_kernel(int atomsCnt, atom* atomList) {

    printf("---------GPU-SINGLE-KERNEL---------\n");
    query_results* res = (query_results*) malloc(sizeof(query_results));
    res->mass = 0;
    res->charge = 0;
    res->max_x = 0;
    res->max_y = 0;
    res->max_z = 0;

    struct timezone i_dunno;
    struct timeval start_time;
    hipStream_t streamComp;

    int* g_s_atomsCnt;
    atom* g_s_atom_list;
    query_results* g_s_res;

    gettimeofday(&start_time, &i_dunno);

    hipStreamCreate(&streamComp);

    hipMalloc((void**)&g_s_res, sizeof(query_results));
    hipMalloc((void**)&g_s_atom_list, sizeof(atom) * atomsCnt);
    hipMalloc((void**)&g_s_atomsCnt, sizeof(int));

    hipMemcpy(g_s_res, res, sizeof(query_results), hipMemcpyHostToDevice);
    hipMemcpy(g_s_atomsCnt, &atomsCnt, sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(g_s_atom_list, atomList, sizeof(atom) * atomsCnt, hipMemcpyHostToDevice);
    hipStreamSynchronize(streamComp);

    /**
    * KERNEL CALL
    */
    int blockSize = 1024;
    int gridSize = ceil(atomsCnt / (float)blockSize) + 1;
    //int stripe = 1024 / ;

    int sizeOfSharedMem = sizeof(float) * gridSize;
    gpu_single_kernel<<<1, gridSize, sizeOfSharedMem, streamComp >>>(g_s_atomsCnt, g_s_atom_list, g_s_res);
    
    hipStreamSynchronize(streamComp);

    /**
    * DATA COPY TO HOST
    */
    hipMemcpy(res, g_s_res, sizeof(query_results), hipMemcpyDeviceToHost);

    float elapsed = time_calc(start_time); 
    printf("%-40s %.3fsec\n", "Running time: ", elapsed / 1000.0f);

    /**
    * MEM FREE
    */
    hipFree(g_s_atom_list);
    hipFree(g_s_res);
    hipFree(g_s_atomsCnt);
}