#include "hip/hip_runtime.h"
#include "gpu_single.h"
#include "types.h"

//added grid size 2 dimensions

//1 body functions
__global__
void gpu_one_body_functions_kernel(int* g_s_atomsCnt, atom* g_s_atom_list, query_results* g_s_res) {

    extern __shared__ int sdata[];
    
    long index_x = blockIdx.x * blockDim.x + threadIdx.x;
    long index_y = blockIdx.y * blockDim.y + threadIdx.y;

    // map the two 2D indices to a single linear, 1D index
    long grid_width = gridDim.x * blockDim.x;
    long i = index_y * grid_width + index_x;

    if(i >= *g_s_atomsCnt) {
        return;
    }

    //for some reason shared memory is becoming slower
    // //shared memory structure:

    // //first mass
    // sdata[tid] = g_s_atom_list[i].mass;
    // //second charge
    // sdata[blockDim.x + tid] = g_s_atom_list[i].charge;

    // while(i < *g_s_atomsCnt) {
    //     sdata[tid] += g_s_atom_list[i].mass;
    //     sdata[blockDim.x + tid] += g_s_atom_list[i].charge;

    //     i += blockDim.x;

    //     __syncthreads();
    // }

    // atomicAdd(&g_s_res->mass, sdata[tid]);
    // atomicAdd(&g_s_res->charge, sdata[blockDim.x + tid]);

    //current atom instance
    atom atomInstance = g_s_atom_list[i];

    atomicAdd(&g_s_res->mass, atomInstance.mass);
    atomicAdd(&g_s_res->charge, atomInstance.charge);

    atomicAdd(&g_s_res->inertiaX, (double)(atomInstance.mass * atomInstance.x));
    atomicAdd(&g_s_res->inertiaY, (double)(atomInstance.mass * atomInstance.y));
    atomicAdd(&g_s_res->inertiaZ, (double)(atomInstance.mass * atomInstance.z));
}

//2 body functions (SDH or POINT DISTANCE HISTOGRAM)
__global__
void gpu_two_body_functions_kernel(atom* at_list, int PDH_acnt, bucket* hist, int num_buckets, double PDH_res) {

    extern __shared__ unsigned long long smem[];

    unsigned long long* shared_histo = smem;
    atom* sharedAtoms = (atom*) &shared_histo[num_buckets];

    long index_x = blockIdx.x * blockDim.x + threadIdx.x;
    long index_y = blockIdx.y * blockDim.y + threadIdx.y;

    // map the two 2D indices to a single linear, 1D index
    long grid_width = gridDim.x * blockDim.x;
    long index = index_y * grid_width + index_x;

    long i = index;

    //check the bound
    if(i >= PDH_acnt) {
        return;
    }

    //for every first thread of the block
    if(threadIdx.x == 0) {
        for(i = 0; i < num_buckets; i++) {
            shared_histo[i] = 0;
        }

        int start = index;
        int k = 0;
        for(i = start; i < start + blockDim.x && i < PDH_acnt; i++, k++) {
            sharedAtoms[k] = at_list[i];
        }
    }

    __syncthreads();

    i = index;
    
    int threadLoad = (PDH_acnt + 1) / 2;

    int start = i + 1;
    int end = i + threadLoad;

    if(PDH_acnt % 2 == 0 && i < PDH_acnt / 2) {
        end++;
    }

    int bi = blockDim.x * blockIdx.x;   // block start
    int ei = bi + blockDim.x;           // block end

    int ind1 = threadIdx.x;             // in this block from sharedAtoms
    int ind2;

    int j;
    for(j = start; j < end; j++) {

        ind2 = j % PDH_acnt;

        double x1 = sharedAtoms[ind1].x;
        double y1 = sharedAtoms[ind1].y;
        double z1 = sharedAtoms[ind1].z;

        double x2, y2, z2;
        
        if(bi <= ind2 && ind2 < ei) {
            x2 = sharedAtoms[ind2 - bi].x;
            y2 = sharedAtoms[ind2 - bi].y;
            z2 = sharedAtoms[ind2 - bi].z;
        } else {
            x2 = at_list[ind2].x;
            y2 = at_list[ind2].y;
            z2 = at_list[ind2].z;
        }

        double dist = sqrt((x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) + (z1 - z2) * (z1 - z2));
        int h_pos = (int) (dist / PDH_res);
        atomicAdd(&shared_histo[h_pos], 1);
    }

    __syncthreads();

    if(threadIdx.x == 0) {
        for(i = 0; i < num_buckets; i++) {
            atomicAdd(&hist[i].d_cnt, shared_histo[i]);
        }
    }
}

void output_histogram(bucket* hist, int num_buckets){
    int i; 
    unsigned long long total_cnt = 0;
    for(i = 0; i < num_buckets; i++) {
        if(i % 5 == 0) /* we print 5 buckets in a row */
            printf("\n%02d: ", i);
        printf("%15lld ", hist[i].d_cnt);
        total_cnt += hist[i].d_cnt;
        /* we also want to make sure the total distance count is correct */
        if(i == num_buckets - 1)    
            printf("\nT:%lld \n", total_cnt);
        else printf("| ");
    }
}


void run_single_kernel(int atomsCnt, atom* atomList) {

    printf("---------GPU-SINGLE-KERNEL---------\n");

    int BOX_SIZE = 175;
    int PDH_res = 1;

    int num_buckets = BOX_SIZE + 1;

    query_results* res = (query_results*) malloc(sizeof(query_results));
    bucket* histogram = (bucket *)malloc(sizeof(bucket) * num_buckets); 
    
    //set default empty values to remove some garbage inside
    res->mass = 0;
    res->charge = 0;
    res->max_x = 0;
    res->max_y = 0;
    res->max_z = 0;
    res->inertiaX = 0;
    res->inertiaY = 0;
    res->inertiaZ = 0;

    int i;
    for(i = 0; i < num_buckets; i++) {
        histogram[i].d_cnt = 0;
    }

    struct timezone i_dunno;
    struct timeval start_time;
    hipStream_t streamComp1, streamComp2;

    //Device Types
    int* g_s_atomsCnt;
    atom* g_s_atom_list;
    bucket* d_histogram;
    query_results* g_s_res;

    gettimeofday(&start_time, &i_dunno);

    hipStreamCreate(&streamComp1);
    hipStreamCreate(&streamComp2);

    hipMalloc((void**)&g_s_res, sizeof(query_results));
    hipMalloc((void**)&g_s_atom_list, sizeof(atom) * atomsCnt);
    hipMalloc((void**)&g_s_atomsCnt, sizeof(int));
    hipMalloc((void**)&d_histogram, num_buckets * sizeof(bucket));

    hipMemcpy(g_s_res, res, sizeof(query_results), hipMemcpyHostToDevice);
    hipMemcpy(g_s_atomsCnt, &atomsCnt, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_histogram, histogram, num_buckets * sizeof(bucket), hipMemcpyHostToDevice);
    hipMemcpy(g_s_atom_list, atomList, sizeof(atom) * atomsCnt, hipMemcpyHostToDevice);

    
    dim3 block_size;
    //static sizes due to big volume of data
    block_size.x = 1024;
    block_size.y = 1;

    // configure a two dimensional grid as well
    dim3 grid_size;

    int maxGridX = 64000;
    if(atomsCnt < block_size.x * maxGridX) {
        grid_size.x = atomsCnt / block_size.x + 1;
        grid_size.y = 1;
    } else {
        grid_size.x = maxGridX;
        grid_size.y = atomsCnt / (block_size.x * maxGridX);
    }

    printf("grid_size.x/y: %d/%d\n", grid_size.x, grid_size.y);

    //int blockSize = 1024;
    //int gridSize = ceil(atomsCnt / (float)blockSize) + 1;
    //int stripe = 1024 / ;

    /**
    * KERNEL CALLS
    */
    //mass and charge
    //----------------------------------1 BODY KERNEL---------------------------------------------------
    int smem1 = sizeof(float) * block_size.x * 2; //this is not really used for now
    gpu_one_body_functions_kernel<<<grid_size, block_size, smem1, streamComp1 >>>(g_s_atomsCnt, g_s_atom_list, g_s_res);

    //----------------------------------2 BODY KERNEL---------------------------------------------------
    int smem2 = num_buckets * sizeof(unsigned long long) + block_size.x * sizeof(atom);
    gpu_two_body_functions_kernel<<<grid_size, block_size, smem2, streamComp2 >>>(g_s_atom_list, atomsCnt, d_histogram, num_buckets, PDH_res);
    
    hipStreamSynchronize(streamComp1);
    hipStreamSynchronize(streamComp2);

    /**
    * DATA COPY TO HOST
    */
    hipMemcpy(res, g_s_res, sizeof(query_results), hipMemcpyDeviceToHost);
    hipMemcpy(histogram, d_histogram, num_buckets * sizeof(bucket), hipMemcpyDeviceToHost);

    float elapsed = time_calc(start_time); 
    printf("%-40s %.3f\n", "Mass Result: ", res->mass);
    printf("%-40s %.3f\n", "Charge Result: ", res->charge);
    printf("%-40s %.3f\n", "Inertia X Axis: ", res->inertiaX);
    printf("%-40s %.3f\n", "Inertia Y Axis: ", res->inertiaY);
    printf("%-40s %.3f\n", "Inertia Z Axis: ", res->inertiaZ);
    printf("%-40s %.3fmillis\n", "Running time: ", elapsed);
    output_histogram(histogram, num_buckets);


    /**
    * MEM FREE
    */
    hipFree(g_s_atom_list);
    hipFree(g_s_res);
    hipFree(g_s_atomsCnt);
}