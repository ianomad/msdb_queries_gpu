#include "hip/hip_runtime.h"
#include "gpu_single.h"
#include "types.h"

__global__ void gpu_single_kernel(int* g_s_atoms_cnt, atom* g_s_atom_list, query_results* g_s_res) {

    extern __shared__ int sdata[];

    int tid = threadIdx.x;

    if(tid >= *g_s_atoms_cnt) {
        return;
    }
    
    int i = tid + blockDim.x;
    sdata[tid] = g_s_atom_list[tid].mass;

    while(i < *g_s_atoms_cnt) {
        sdata[tid] += g_s_atom_list[i].mass;
        i += blockDim.x;
        __syncthreads();
    }

    atomicAdd(&g_s_res->mass, sdata[tid]);
}

void run_single_kernel(int atoms_cnt, int workload) {

    printf("---------GPU-SINGLE-KERNEL---------\n");
    atom* atom_list = (atom*)malloc(sizeof(atom) * atoms_cnt);

    query_results* res = (query_results*) malloc(sizeof(query_results));
    res->mass = 0;

    int w;
    int block_size = 1024;
    struct timezone i_dunno;
    struct timeval start_time;
    hipStream_t streamComp;
    hipStream_t streamCpy;

    hipEvent_t start, stop;
    float elapsedTime;

    int* g_s_atoms_cnt;
    atom* g_s_atom_list;
    atom* g_s_atom_list_pinned;
    query_results* g_s_res;

    gettimeofday(&start_time, &i_dunno);

    hipStreamCreate(&streamComp);
    hipStreamCreate(&streamCpy);

    hipMalloc((void**)&g_s_res, sizeof(query_results));
    hipMalloc((void**)&g_s_atom_list, sizeof(atom) * atoms_cnt);
    hipMalloc((void**)&g_s_atom_list_pinned, sizeof(atom) * atoms_cnt);
    hipMalloc((void**)&g_s_atoms_cnt, sizeof(int));

    hipMemcpy(g_s_res, res, sizeof(query_results), hipMemcpyHostToDevice);
    hipMemcpy(g_s_atoms_cnt, &atoms_cnt, sizeof(int), hipMemcpyHostToDevice);

    hipEventCreate(&start);
    hipEventRecord(start, 0);

    generate_data(atom_list, atoms_cnt);

    for(w = 0; w < workload; w++) {
        
        hipMemcpyAsync(g_s_atom_list, atom_list, sizeof(atom) * atoms_cnt, hipMemcpyHostToDevice, streamCpy);
        
        hipStreamSynchronize(streamCpy);
        hipStreamSynchronize(streamComp);

        /**
        * KERNEL CALL
        */
        int grid_size = ceil(atoms_cnt / (float)block_size) + 1;// + (atoms_cnt % block_size == 0 ? 0 : 1);
        int stripe = 1024;
        gpu_single_kernel<<<1, stripe, sizeof(float) * stripe, streamComp >>>(g_s_atoms_cnt, g_s_atom_list, g_s_res);

        atom* tmp = g_s_atom_list;
        g_s_atom_list = g_s_atom_list_pinned;
        g_s_atom_list_pinned = tmp;
    }
    
    hipStreamSynchronize(streamComp);

    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    /**
    * DATA COPY TO HOST
    */
    hipMemcpy(res, g_s_res, sizeof(query_results), hipMemcpyDeviceToHost);

    printf("%-40s %.2f\n", "Sum of masses:", res->mass);
    //float elapsed = time_calc(start_time); 
    printf("%-40s %.3fsec\n", "Running time: ", elapsedTime / 1000.0f);

    /**
    * MEM FREE
    */
    hipFree(g_s_atom_list);
    hipFree(g_s_res);
    hipFree(g_s_atoms_cnt);
}